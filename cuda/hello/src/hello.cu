// Hello World in CUDA !


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
  printf("Hello World from CUDA !\n");
}

int main() {
  hello<<<1, 1>>>();
  hipDeviceSynchronize();
  return 0;
}
